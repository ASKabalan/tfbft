#include "hip/hip_runtime.h"

#include "common/ffi_helper.hpp"
#include <cute/tensor.hpp>
#include "xla/ffi/api/api.h"
#include "xla/ffi/api/ffi.h"
#include <cute/numeric/complex.hpp>

namespace ffi = xla::ffi;
using namespace cute;
template <typename T>
using cuteComplex = cutlass::complex<T>;

template <class TensorS, class ThreadLayout>
__global__ void bit_reverse_kernel(TensorS S, ThreadLayout) {
    using Element = typename TensorS::value_type;

    unsigned int tid = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int tid_y = threadIdx.y + blockIdx.y * blockDim.y;
    unsigned int tid_z = threadIdx.z + blockIdx.z * blockDim.z;
    if (tid >= size<0>(S) || tid_y >= size<1>(S) || tid_z >= size<2>(S)) return;

    unsigned int bit_reversed_tid = __brev(tid) >> (__clz(size<0>(S)) + 1);
    unsigned int bit_reversed_blockIdx = bit_reversed_tid / (unsigned int)blockDim.x;
    unsigned int bit_reversed_threadIdx = bit_reversed_tid % (unsigned int)blockDim.x;

    if (tid < bit_reversed_tid) return;

    if (tid == bit_reversed_tid && blockIdx.x == bit_reversed_blockIdx) return;

    Tensor natural_order_tile = S(make_coord(_, _, _), blockIdx.x, blockIdx.y, blockIdx.z);
    Tensor reversed_order_tile = S(make_coord(_, _, _), bit_reversed_blockIdx, blockIdx.y, blockIdx.z);

    auto natural_order_threadid = threadIdx.x + threadIdx.y * stride<1>(ThreadLayout{}) +
                                  threadIdx.z * stride<2>(ThreadLayout{}) * stride<1>(ThreadLayout{});
    auto bit_reversed_threadid = bit_reversed_threadIdx + threadIdx.y * stride<1>(ThreadLayout{}) +
                                 threadIdx.z * stride<2>(ThreadLayout{}) * stride<1>(ThreadLayout{});

    Tensor natural_order_thr = local_partition(natural_order_tile, ThreadLayout{}, natural_order_threadid);
    Tensor reversed_order_thr = local_partition(reversed_order_tile, ThreadLayout{}, bit_reversed_threadid);
    // Copy from GMEM to RMEM
    Tensor rmem_tensor = make_fragment_like(natural_order_thr);

    copy(natural_order_thr, rmem_tensor);
    copy(reversed_order_thr, natural_order_thr);
    copy(rmem_tensor, reversed_order_thr);
}

template <class TensorS>
ffi::Error bit_reverse_impl(hipStream_t stream, TensorS tensor_S) {
    auto block_shape = Shape<_64, _4, _4>{};
    auto thread_layout = make_layout(Shape<_64, _4, _4>{});
    Tensor tiled_tensor_S = tiled_divide(tensor_S, block_shape);

    dim3 gridDim(size<1>(tiled_tensor_S), size<2>(tiled_tensor_S), size<3>(tiled_tensor_S));
    dim3 blockDim(shape<0>(thread_layout), shape<1>(thread_layout), shape<2>(thread_layout));

    bit_reverse_kernel<<<gridDim, blockDim, 0, stream>>>(tiled_tensor_S, thread_layout);
    return ffi_with_cuda_error_check();
}

template <ffi::DataType dtype>
ffi::Error bit_reverse_complex_tensor(hipStream_t stream, Buffer<dtype> x, Result<dtype> y) {
    static_assert(ffi::IsComplexType<dtype>(), "Only complex types in this function");
    using Real = ffi::NativeType<ffi::ToReal(dtype)>;
    using Complex = cuteComplex<Real>;

    auto dimensions = x.dimensions();
    auto shape = make_shape(dimensions[0], dimensions[1], dimensions[2]);
    auto ptr_D = reinterpret_cast<Complex*>(y->typed_data());
    Tensor tensor_D = make_tensor(make_gmem_ptr(ptr_D), make_layout(shape));

    return bit_reverse_impl(stream, tensor_D);
}

template <ffi::DataType dtype>
ffi::Error bit_reverse_real_tensor(hipStream_t stream, Buffer<dtype> x, Result<dtype> y) {
    static_assert(!ffi::IsComplexType<dtype>(), "Only real types in this function");

    auto dimensions = x.dimensions();
    auto shape = make_shape(dimensions[0], dimensions[1], dimensions[2]);
    Tensor tensor_D = make_tensor(make_gmem_ptr(y->typed_data()), make_layout(shape));

    return bit_reverse_impl(stream, tensor_D);
}

XLA_FFI_DEFINE_HANDLER_SYMBOL(BitReverseHandlerC64, bit_reverse_complex_tensor<DataType::C64>,
                              ffi::Ffi::Bind()
                                      .Ctx<FFI_Stream_Type>()
                                      .Arg<Buffer<DataType::C64>>()  // x
                                      .Ret<Buffer<DataType::C64>>()  // y
);

XLA_FFI_DEFINE_HANDLER_SYMBOL(BitReverseHandlerC128, bit_reverse_complex_tensor<DataType::C128>,
                              ffi::Ffi::Bind()
                                      .Ctx<FFI_Stream_Type>()
                                      .Arg<Buffer<DataType::C128>>()  // x
                                      .Ret<Buffer<DataType::C128>>()  // y
);

XLA_FFI_DEFINE_HANDLER_SYMBOL(BitReverseHandlerF32, bit_reverse_real_tensor<DataType::F32>,
                              ffi::Ffi::Bind()
                                      .Ctx<FFI_Stream_Type>()
                                      .Arg<Buffer<DataType::F32>>()  // x
                                      .Ret<Buffer<DataType::F32>>()  // y
);

XLA_FFI_DEFINE_HANDLER_SYMBOL(BitReverseHandlerF64, bit_reverse_real_tensor<DataType::F64>,
                              ffi::Ffi::Bind()
                                      .Ctx<FFI_Stream_Type>()
                                      .Arg<Buffer<DataType::F64>>()  // x
                                      .Ret<Buffer<DataType::F64>>()  // y
);
