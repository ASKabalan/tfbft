#include "hip/hip_runtime.h"
#include "gpu_ops.h"
#include <hip/hip_runtime.h>

__global__ void add_element_kernel(float *x, float *y, int n) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < n) {
    y[i] = x[i] * 2;
  }
}

void add_element(float *x, float *y, int n, hipStream_t stream) {
  int block_size = 256;
  int num_blocks = (n + block_size - 1) / block_size;
  add_element_kernel<<<num_blocks, block_size, 0, stream>>>(x, y, n);
}

