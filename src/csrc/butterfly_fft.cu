#include "hip/hip_runtime.h"
#include "collectives/butterfly_comm_index.hpp"
#include "collectives/collective_ops.hpp"
#include "common/ffi_helper.hpp"
#include "extensions.h"
#include "matx.h"
#include "matx/core/make_tensor.h"
#include "xla/ffi/api/api.h"
#include "xla/ffi/api/ffi.h"
#include <cstddef>
#include <cstdint>
#include <hip/hip_runtime.h>
#include <nccl.h>

namespace ffi = xla::ffi;

template <typename T> __global__ void AddElementKernel(T *x, T *y, int n) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < n) {
    y[i] = x[i] * 2;
  }
}

template <DataType dtype>
ffi::Error ButterFlyFFT(hipStream_t stream, Buffer<DataType::F32> x,
                        int64_t iDirection, int64_t iAxis, int64_t iNorm,
                        Result<dtype> y) {

  CCO::CollectiveOps ops;
  AXIS axis = static_cast<AXIS>(iAxis);
  DIRECTION direction = static_cast<DIRECTION>(iDirection);
  NORM norm = static_cast<NORM>(iNorm);

  assertm(axis == AXIS::X, "Only X axis is supported");

  ncclComm_t comm = CCO::NCCLOps::get_comm();
  const int &rank = CCO::NCCLOps::get_rank();
  const int &size = CCO::NCCLOps::get_size();
  matx::index_t dims[3]{1, 1, 1};
  Dimensions buffer_dims = x.dimensions();
  std::copy(buffer_dims.begin(), buffer_dims.end(), dims);

  auto butterfly_comm = ButterflyCommIndex::get_or_create_comms(comm);
  auto tensor_s = matx::make_tensor(x.typed_data(), dims);
  auto tensor_d = matx::make_tensor(y->typed_data(), dims);
  CCO::ReductionOp reduc_op{CCO::ReducType::SUM};
  
  auto comm_iter = 

  for (ncclComm_t comm : butterfly_comm) {
    ops.allreduce(x.typed_data(), y->typed_data(), x.element_count(), reduc_op,
                  comm, stream);
  }

  return ffi_with_cuda_error_check();
}

XLA_FFI_DEFINE_HANDLER_SYMBOL(ButterFlyFFTHandlerF32,
                              ButterFlyFFT<DataType::F32>,
                              ffi::Ffi::Bind()
                                  .Ctx<FFI_Stream_Type>()
                                  .Arg<Buffer<DataType::F32>>() // x
                                  .Attr<int64_t>("direction")
                                  .Attr<int64_t>("norm")
                                  .Attr<int64_t>("axis")
                                  .Ret<Buffer<DataType::F32>>() // y
);
