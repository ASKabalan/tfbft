#include "hip/hip_runtime.h"
#include "collectives/butterfly_comm_index.hpp"
#include "collectives/collective_ops.hpp"
#include "common/ffi_helper.hpp"
#include "extensions.h"
#include "matx.h"
#include "matx/core/make_tensor.h"
#include "xla/ffi/api/api.h"
#include "xla/ffi/api/ffi.h"
#include <cstddef>
#include <cstdint>
#include <hip/hip_runtime.h>
#include <nccl.h>

namespace ffi = xla::ffi;

template <typename T> __global__ void AddElementKernel(T *x, T *y, int n) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < n) {
    y[i] = x[i] * 2;
  }
}

template <DataType dtype>
ffi::Error ButterFlyForward(hipStream_t stream, Buffer<DataType::F32> x,
                            NORM iNorm, Result<dtype> y, CommIterator &comms) {

  CCO::CollectiveOps ops;
  CCO::ReductionOp reduc_op{CCO::ReducType::SUM};
  auto comm = comms.next();
  assertm(comm.has_value(),
          "[INTERNAL] Distributed FFT called without Buttefly comms");

  ops.allreduce(x.typed_data(), y->typed_data(), x.element_count(), reduc_op,
                comm.value(), stream);

  while ((comm = comms.next())) {
    ops.allreduce(y->typed_data(), y->typed_data(), x.element_count(), reduc_op,
                  comm.value(), stream);
  }
  return ffi_with_cuda_error_check();
}

template <DataType dtype>
ffi::Error ButterFlyBackward(hipStream_t stream, Buffer<DataType::F32> x,
                             NORM iNorm, Result<dtype> y, CommIterator &comms) {

  CCO::CollectiveOps ops;
  CCO::ReductionOp reduc_op{CCO::ReducType::SUM};
  comms.reset();
  auto comm = comms.prev();
  assertm(comm.has_value(),
          "[INTERNAL] Distributed IFFT called without Buttefly comms");

  ops.allreduce(x.typed_data(), y->typed_data(), x.element_count(), reduc_op,
                comm.value(), stream);

  while ((comm = comms.prev())) {
    ops.allreduce(y->typed_data(), y->typed_data(), x.element_count(), reduc_op,
                  comm.value(), stream);
  }
  return ffi_with_cuda_error_check();
}

template <DataType dtype>
ffi::Error ButterFlyFFT(hipStream_t stream, Buffer<DataType::F32> x,
                        int64_t iDirection, int64_t iAxis, int64_t iNorm,
                        Result<dtype> y) {

  AXIS axis = static_cast<AXIS>(iAxis);
  DIRECTION direction = static_cast<DIRECTION>(iDirection);
  NORM norm = static_cast<NORM>(iNorm);

  assertm(axis == AXIS::X, "Only X axis is supported");

  ncclComm_t comm = CCO::NCCLOps::get_comm();
  const int &rank = CCO::NCCLOps::get_rank();
  const int &size = CCO::NCCLOps::get_size();
  matx::index_t dims[3]{1, 1, 1};
  Dimensions buffer_dims = x.dimensions();
  std::copy(buffer_dims.begin(), buffer_dims.end(), dims);

  auto butterfly_comm = ButterflyCommIndex::get_or_create_comms(comm);
  auto tensor_s = matx::make_tensor(x.typed_data(), dims);
  auto tensor_d = matx::make_tensor(y->typed_data(), dims);
  CCO::ReductionOp reduc_op{CCO::ReducType::SUM};

  switch (direction) {
  case DIRECTION::FORWARD:
    return ButterFlyForward(stream, x, norm, y, butterfly_comm);
  case DIRECTION::INVERSE:
    return ButterFlyBackward(stream, x, norm, y, butterfly_comm);
  default:
    return ffi::Error(XLA_FFI_Error_Code_INTERNAL,
                      std::string("Un recongnized FFT direction "));
  }
}

XLA_FFI_DEFINE_HANDLER_SYMBOL(ButterFlyFFTHandlerF32,
                              ButterFlyFFT<DataType::F32>,
                              ffi::Ffi::Bind()
                                  .Ctx<FFI_Stream_Type>()
                                  .Arg<Buffer<DataType::F32>>() // x
                                  .Attr<int64_t>("direction")
                                  .Attr<int64_t>("norm")
                                  .Attr<int64_t>("axis")
                                  .Ret<Buffer<DataType::F32>>() // y
);
